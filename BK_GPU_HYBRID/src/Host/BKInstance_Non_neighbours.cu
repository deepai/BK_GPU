#include "BKInstance.h"
#include "../moderngpu/util/mgpucontext.h"
#include "../kernels/kernels.cuh"
#include "../moderngpu/mgpuhost.cuh"
#include "../cub/device/device_scan.cuh"

using namespace mgpu;

namespace BK_GPU {

/**
 * Requires: Updated TopElement,Updated Stack,Sorted P and X segments.
 *
 * This element finds the next non_neighbour of the pivot.Once the non_neighbour is located it is moved to the end of the current P segment.
 * Thus R segment is increased by 1 and P segment is reduced by 1. The beginning of R is shifted by 1 towards P.
 *
 * At the end of this method,
 * The topElement is updated,stack is updated with the new configuration.
 * In this new configuration,the pivot element is moved to the R segment and the P segment size is changed to
 * only contain the neighbors of Pivot in the P segment. i.e.
 *
 * X segment is updated to contain only the neighbors of the pivot,the current size of X is updated to reflect that.
 * Both P and X segments are sorted post this method.
 *
 * Tracker is updated but contains the previous value itself.
 *
 * @param element
 * @return
 */
void BKInstance::nextNonPivot(int pivot)
{
	int NullValue;
	//Obtain the TopElement
	stack->topElement(&topElement);

	//CudaError(hipStreamSynchronize(*(this->Stream)));

	//AdjacencySize
	int adjacencyPivotSize = host_graph->rowOffset[pivot+1] - host_graph->rowOffset[pivot];

	//Obtain the elements for the bdata
	unsigned int *bdata = gpuGraph->Columns + host_graph->rowOffset[pivot];
	int  bcount = adjacencyPivotSize;

	//obtain the elements for the adata
	unsigned int *adata = (Ng->data) +topElement.beginP;
	int  acount = topElement.currPSize;

	unsigned *ptr;

	size_t requiredSize = sizeof(uint)*(topElement.currPSize);

	//Allocate memory of size 2*currP
	CudaError(hipMalloc(&ptr,requiredSize));

//	//Sort P segment once
//	if(topElement.currPSize > 1)
//		CudaError(hipcub::DeviceRadixSort::SortKeys(ptr,requiredSize,adata,adata,acount,0,sizeof(uint)*8,*(this->Stream)));

	int currNeighbour,non_neighbours;

	//This sorted search is used to know the values which are non-neighbours with respect to pivot.
	//This values are indicated with 0s
	SortedSearch<MgpuBoundsLower, MgpuSearchTypeMatch, MgpuSearchTypeNone>(
					adata, acount, bdata, bcount, ptr, ptr, **Context,
					&currNeighbour, &non_neighbours);

	//Locate and swap the last zeroes.
	GpuArraySwapNonPivot(Ng,ptr,topElement.beginP,topElement.beginP + topElement.currPSize - 1,currNeighbour,*(this->Stream));

	// Check bounds and swap if the selected element is not at beginR - 1 position
	if((topElement.beginP + topElement.currPSize - 1) != (topElement.beginR - 1))
	{
		GpuSwap(Ng,topElement.beginP + topElement.currPSize - 1,topElement.beginR - 1,*(this->Stream));
	}

	unsigned nextCandidateNode; //= Ng->data[topElement.beginR-1];

	CudaError(hipMemcpy(&nextCandidateNode,Ng->data + topElement.beginR - 1 ,sizeof(unsigned ),hipMemcpyDeviceToHost));


	bdata  = (Ng->data) + host_graph->rowOffset[nextCandidateNode];
	bcount = host_graph->rowOffset[nextCandidateNode+1] - host_graph->rowOffset[nextCandidateNode];

	CudaError(hipFree(ptr));

	#pragma omp parallel num_threads(2)
	{
		int threadId = omp_get_thread_num();

		if(threadId == 0)
		{

			hipStream_t currStream = Context[threadId]->Stream();

			int PCount = topElement.currPSize - 1;
			unsigned *Pdata = Ng->data + topElement.beginP;
			unsigned *aux;

			CudaError(hipMalloc(&aux,sizeof(unsigned)*topElement.beginP));

			if(topElement.currPSize > 1)
			{
				//update the size of acount

				//if P segment was greater than 2, then sort the remaining P segment.
				if(topElement.currPSize > 2)
				{
					void *d_temp_storage=NULL;size_t d_temp_size=0;

					CudaError(hipcub::DeviceRadixSort::SortKeys(d_temp_storage,d_temp_size,Pdata,Pdata,PCount,0,sizeof(uint)*8,currStream));

					CudaError(hipMalloc(&d_temp_storage,d_temp_size));

					if(d_temp_storage==NULL)
								d_temp_storage=&NullValue;

					CudaError(hipcub::DeviceRadixSort::SortKeys(d_temp_storage,d_temp_size,Pdata,Pdata,PCount,0,sizeof(uint)*8,currStream));

					CudaError(hipStreamSynchronize(currStream));

					if(d_temp_storage!=&NullValue)
						CudaError(hipFree(d_temp_storage));
				}
			}

			//Intersection of currP with the neighbors of nextCandidateNode
			SortedSearch<MgpuBoundsLower, MgpuSearchTypeMatch, MgpuSearchTypeNone>(
								Pdata, PCount, bdata, bcount, aux, aux, *(Context[threadId]),
								&currNeighbour, &non_neighbours);

			CudaError(hipStreamSynchronize(currStream));

			topElement.currPSize = currNeighbour;

			//Do an Inclusive Scan on the intersection values of the adata
			if(topElement.currPSize > 2)
			{
				void *d_temp_storage=NULL;size_t d_temp_size=0;

				//Ist Invocation calculates the amount of memory required for the temporary array.
				CudaError(hipcub::DeviceScan::InclusiveSum(d_temp_storage,d_temp_size,aux,aux,topElement.currPSize - 1,currStream));

				CudaError(hipMalloc(&d_temp_storage,d_temp_size));

				//This step does the actual inclusiveSum
				CudaError(hipcub::DeviceScan::InclusiveSum(d_temp_storage,d_temp_size,aux,aux,topElement.currPSize - 1,currStream));

				CudaError(hipStreamSynchronize(currStream));

				if(d_temp_storage!=&NullValue)
					CudaError(hipFree(d_temp_storage));

			}

			//Non_neighbour of the current selected candidate Vertex
			non_neighbours = topElement.currPSize - 1 - currNeighbour;

			//if size of neighbors is atleast 1 and less than currPSize
			if((currNeighbour>0) && (currNeighbour < (topElement.currPSize - 1)))
			{
				GpuArrayRearrangeP(this->Ng, this->stack, this->gpuGraph, aux,
					topElement.beginP, topElement.beginP + topElement.currPSize - 2,non_neighbours,currStream);
			}

		}
		else
		{
			hipStream_t currStream = Context[threadId]->Stream();

			int XCount = topElement.currXSize;
			unsigned *Xdata = Ng->data + topElement.beginX;
			unsigned *auxX;

			CudaError(hipMalloc(&auxX,sizeof(int)*topElement.currXSize));

			if(topElement.currXSize!=0)
			{

				int NeighboursinX, nonNeighboursinX;

				SortedSearch<MgpuBoundsLower, MgpuSearchTypeMatch, MgpuSearchTypeNone>(
								Xdata, XCount, bdata, bcount, auxX,auxX, **Context,
								&NeighboursinX, &nonNeighboursinX);

				CudaError(hipStreamSynchronize(currStream));

				if(topElement.currXSize > 1)
				{
					/***
					 * * Do a Scan on the current dptr array. We can use the prefix sum to rearrange the neighbours and non-neighbours
					 */		//thrust::inclusive_scan(dptr, dptr + currX, dptr);
					void *d_temp_storage=NULL;size_t d_temp_size=0;

					CudaError(hipcub::DeviceScan::InclusiveSum(d_temp_storage,d_temp_size,auxX,auxX,topElement.currXSize,currStream));

					CudaError(hipMalloc(&d_temp_storage,d_temp_size));

					CudaError(hipcub::DeviceScan::InclusiveSum(d_temp_storage,d_temp_size,auxX,auxX,topElement.currXSize,currStream));

					CudaError(hipStreamSynchronize(currStream));

					if(d_temp_storage!=&NullValue)
						CudaError(hipFree(d_temp_storage));
				}

				nonNeighboursinX = topElement.currXSize - NeighboursinX;

				if((NeighboursinX > 0) && (NeighboursinX < topElement.currXSize ))
					GpuArrayRearrangeX(Ng,stack,gpuGraph,auxX,topElement.beginX,topElement.beginX + topElement.currXSize - 1,nonNeighboursinX,currStream);

				topElement.currXSize = NeighboursinX;

			}
		}
	}

	topElement.beginR = topElement.beginR - 1;
	topElement.currRSize = topElement.currRSize + 1;
	topElement.pivot = nextCandidateNode;
	topElement.direction = true;

	//CudaError(hipStreamSynchronize(*(this->Stream)));

	stack->push(&topElement);

		//CudaError(hipStreamSynchronize(*(this->Stream)));

	}



}


