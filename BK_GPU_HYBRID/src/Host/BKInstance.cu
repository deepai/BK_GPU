/*
 * BKInstance.cpp
 *
 *  Created on: 06-Oct-2015
 *      Author: debarshi
 */

#include "BKInstance.h"
#include "../moderngpu/util/mgpucontext.h"
#include "../kernels/kernels.cuh"
#include "../moderngpu/mgpuhost.cuh"
#include "../cub/device/device_scan.cuh"

using namespace mgpu;

namespace BK_GPU {

BKInstance::BKInstance(Graph *host_graph, BK_GPU::GPU_CSR *gpuGraph,
		BK_GPU::NeighbourGraph *Ng, BK_GPU::GPU_Stack *stack,hipStream_t &stream) {
	// TODO Auto-generated constructor stub
	this->Ng = Ng;
	this->gpuGraph = gpuGraph;
	this->stack = stack;
	this->topElement = stack->topElement();
	this->hostGraph = new NeighbourGraph();
	this->hostGraph = Ng;

	this->Stream= &stream;

	this->Context = mgpu::CreateCudaDeviceAttachStream(0,*(this->Stream));
	this->host_graph = host_graph;

	this->tracker = new BK_GPU::RecursionStack(topElement.currPSize);
}

int BKInstance::processPivot(BK_GPU::StackElement &element) {
	/**Step 1: Find the pivot element
	 */
	int currP = topElement.currPSize; //Size of Number of Elements in P
	int currX = topElement.currXSize;
	unsigned int *d_Sorted; //This is used to store the Unsorted elements initially

	void *d_temp_storage = NULL; //Auxillary array required for temporary Storage
	size_t d_temp_size = sizeof(unsigned) * currP * 2; //size of auxillary array is 2*N

	//Allocate Auxillary Array
	gpuErrchk(hipMallocManaged(&d_temp_storage, sizeof(unsigned)* 2 * (currP + currX)));

	//Point to the unsorted input data
	unsigned int *d_unSorted = (unsigned *) &(Ng->data[topElement.beginP]);
	d_Sorted = d_unSorted;

	DEV_SYNC
	;
	//Store the Node Value for each value in the currPArray
	unsigned int *hptr = new unsigned[currP];

	//Kernel to copy the current P Values to the host in the hptr array.
	GpuCopyOffsetAddresses(Ng, stack, gpuGraph, hptr, currP,*(this->Stream));

	//This Array contains values 0 and 1 to store whether a value in the needle matches the haystack
	unsigned int* dptr;

	//size currP to allow prefixSums
	gpuErrchk(hipMallocManaged(&dptr, sizeof(int) * 2 *(currP + currX)));

	DEV_SYNC
	;

	unsigned int *adata = d_Sorted;

	/** Max Index is used to store the index of value within P
	 *  Max Index lies between 0 and P-1.
	 */
	int max_index, numNeighbours = -1;

	int currNeighbour, non_neighbours;
	int acount = currP;

	/** For each value in the P array. Obtain the count of its neighbour amongst P.
	 *  The value with the maximum neighbour count is selected as the pivot. Other non-neighbours are also
	 *  selected after the pivot is completed.
	 *  This helps avoid unnecessary computations.
	 *
	 */
	for (int i = 0; i < currP; i++) {
		int adjacencySize = (host_graph->rowOffset[hptr[i] + 1] - host_graph->rowOffset[hptr[i]]);

		//std::cout << adjacencySize << ", " << host_graph->rowOffset[hptr[i] + 1] << " " << host_graph->rowOffset[hptr[i]]<< std::endl;

		unsigned int *bdata =
				&(gpuGraph->Columns[host_graph->rowOffset[hptr[i]]]);

		DEV_SYNC
		; //

		SortedSearch<MgpuBoundsLower, MgpuSearchTypeMatch, MgpuSearchTypeNone>(
				adata, acount, bdata, adjacencySize, dptr, dptr, *Context,
				&currNeighbour, &non_neighbours);


		if (currNeighbour > numNeighbours) {
			max_index = i;
			numNeighbours = currNeighbour;
		}
	}

	/**Swap the element(pivot) with the rightMost P element.
	 * New Size of P
	 */
	int endP = this->topElement.beginR - 1;

	GpuSwap(this->Ng,max_index+topElement.beginP, endP);

	d_unSorted = d_Sorted;
//
	gpuErrchk(hipcub::DeviceRadixSort::SortKeys(d_temp_storage, d_temp_size,
						d_unSorted, d_Sorted, currP-1,0,sizeof(uint)*8,*(this->Stream)));


	DEV_SYNC;

	int newBeginR = topElement.beginR - 1;
	int newRsize = topElement.currRSize + 1;


	//adjacency size of the neighbour array.
	int adjacencySize = host_graph->rowOffset[hptr[max_index] + 1]
			- host_graph->rowOffset[hptr[max_index]];

	//pointer to the beginning of the adjancy list for the maximum value
	unsigned *bdata =
			&(gpuGraph->Columns[host_graph->rowOffset[hptr[max_index]]]);

	//This calculates the number of remaining non-neighbours of pivot.
	SortedSearch<MgpuBoundsLower, MgpuSearchTypeMatch, MgpuSearchTypeNone>(
			adata, acount - 1, bdata, adjacencySize, dptr, dptr, *Context,
			&currNeighbour, &non_neighbours);

	int newPsize = currNeighbour;

	/**
	 * //Do a Scan on the current dptr array.
	 * //thrust::inclusive_scan(dptr, dptr + currP - 1, dptr);
	 */

	if(currP > 2)
	{
		size_t requiredmemSize;void *ptr=NULL;

		//Ist Invocation calculates the amount of memory required for the temporary array.
		gpuErrchk(hipcub::DeviceScan::InclusiveSum(ptr,requiredmemSize,dptr,dptr,currP - 1,*(this->Stream)));

		gpuErrchk(hipMalloc(&ptr,requiredmemSize));

		//This step does the actual inclusiveSum
		gpuErrchk(hipcub::DeviceScan::InclusiveSum(ptr,requiredmemSize,dptr,dptr,currP - 1,*(this->Stream)));

		gpuErrchk(hipFree(ptr));
	}

	DEV_SYNC;


	non_neighbours = currP - 1 - currNeighbour;

	//call Kernel Here to re-arrange P elements
	if((currNeighbour>0) && (currNeighbour < (currP-1)))
	{
		GpuArrayRearrangeP(this->Ng, this->stack, this->gpuGraph, dptr,
			topElement.beginP, topElement.beginP + currP - 2,non_neighbours,*(this->Stream));
	}

	//Repeat the steps for currX.
	//Intersection with X

	if (currX != 0) 
	{
		d_temp_size = 2 * currX * sizeof(int);

		//Pointer to the CurrX Values
		d_unSorted = (unsigned *) &(Ng->data[topElement.beginX]);
		d_Sorted = d_unSorted;

		//Output CurrX sorted into
		gpuErrchk(
				hipcub::DeviceRadixSort::SortKeys(d_temp_storage, d_temp_size,
						d_unSorted, d_Sorted, currX,0,sizeof(uint)*8,*(this->Stream)));

		adata = d_Sorted;
		int acount = topElement.currXSize;

		int NeighboursinX, nonNeighboursinX;

		SortedSearch<MgpuBoundsLower, MgpuSearchTypeMatch, MgpuSearchTypeNone>(
				adata, acount, bdata, adjacencySize, dptr, dptr, *Context,
				&NeighboursinX, &nonNeighboursinX);


		if(currX > 2)
		{
			/***
			 * * Do a Scan on the current dptr array. We can use the prefix sum to rearrange the neighbours and non-neighbours
			 */		//thrust::inclusive_scan(dptr, dptr + currX, dptr);
			size_t requiredmemSize = 0; void *ptr=NULL;

			gpuErrchk(hipcub::DeviceScan::InclusiveSum(ptr,requiredmemSize,dptr,dptr,currX,*(this->Stream)));

			gpuErrchk(hipMalloc(&ptr,requiredmemSize));

			gpuErrchk(hipcub::DeviceScan::InclusiveSum(ptr,requiredmemSize,dptr,dptr,currX,*(this->Stream)));

			gpuErrchk(hipFree(ptr));

			DEV_SYNC;
		}

		/***
		 * Scan Complete
		 */


		if((NeighboursinX > 0) && (NeighboursinX < currX ))
			GpuArrayRearrangeX(Ng,stack,gpuGraph,dptr,topElement.beginX,topElement.beginP-1,NeighboursinX,*(this->Stream));

		topElement.currXSize = NeighboursinX;
	}
	int trackerSize = tracker->size() ;

	stack->push(topElement.beginX, topElement.currXSize, topElement.beginP,
			newPsize, newBeginR, newRsize, max_index,trackerSize, non_neighbours, true);

	topElement.beginR = newBeginR;
	topElement.currPSize = newPsize;
	topElement.currRSize = newRsize;
	topElement.direction = true;
	topElement.pivot = hptr[max_index];
	topElement.remainingNonNeighbour = non_neighbours;

	//debug(max_index, hptr[max_index], numNeighbours);

	//debug(dptr[0],dptr[1],dptr[2],dptr[3]);

	/**Free the pointers **/
	gpuErrchk(hipFree(d_temp_storage));
	gpuErrchk(hipFree(dptr));


	delete[] hptr;

	return (non_neighbours + 1);

}

void BKInstance::printClique(int CliqueSize,int beginClique)
{
#ifdef PRINTCLIQUES
	for(int i=0;i<CliqueSize;i++)
		printf("%d ",Ng->data[beginClique+i]+1);

	printf("\n");
#endif
}

void BKInstance::RunCliqueFinder(int CliqueId) {

//	//topElement.printconfig();
//	if(topElement.currRSize%50==0)
//		topElement.printconfig();

	if ((topElement.currPSize == topElement.currXSize)
			&& (topElement.currXSize == 0)) {		//Obtained a Clique
		printf("%d) Clique of size %d, found!\n",CliqueId,topElement.currRSize);
		printClique(topElement.currRSize,topElement.beginR);
		return;
	} else if (topElement.currPSize == 0)
	{
		//printf("%d) Already contains a clique\n",CliqueId);
		return; //didn't obtain a Clique
	}
	else {
		int non_neighbours = processPivot(topElement);
		RunCliqueFinder(CliqueId);

		stack->pop();

		tracker->push(topElement.pivot);

		tracker->pop();
	}
}

} /* namespace BK_GPU */
