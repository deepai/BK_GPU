#include "hip/hip_runtime.h"
#include "kernels.cuh"

/**This kernel is used to swap the values and bring the values having 1 in the darray towards the
 * start_offset and the values having 0 towards the end_offset
 * This Kernel acts as a gatherer.
 *
 * @param darray //array containing the prefixSum
 * @param d_temp //array to store the gathered values
 * @param start_offset //startOffset in the graph->data
 * @param end_offset //endoffset
 * @param graph //input graph
 * @param stack //input stack
 */
__global__
void kernelRearrangeGatherX(unsigned int *darray,int *d_temp,int start_offset,int end_offset,int countOnes,BK_GPU::NeighbourGraph *graph,BK_GPU::GPU_Stack* stack)
{
  int tid=threadIdx.x + blockDim.x*blockIdx.x;

  //Exceeds limit hence return
  if(tid+start_offset > end_offset)
    return;

  //get the current prefixsum value
  int currVal=darray[tid];

  //get the next prefixsum value
  int prevVal=(tid==0)?0:darray[tid-1];

  int destination; //store destination here

  if(currVal - prevVal == 0)
    destination = tid - currVal;
  else
    destination = end_offset - (countOnes - currVal) -start_offset;

  //Copy the current Element before swapping
  int currElement=graph->data[tid+start_offset];

  d_temp[destination] = currElement;

}

/**
 * In this kernel elements in the d_temp array are copied back into the graph->data
 * starting from start_offset.
 *
 * @param d_temp //temporary storage
 * @param start_offset //start_offset in graph->data
 * @param end_offset //end_offset in graph->data
 * @param graph //graph
 */
__global__
void KernelRearrangeScatterX(int *d_temp,int start_offset,int end_offset,BK_GPU::NeighbourGraph *graph)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if(tid+start_offset > end_offset)
    return;

  graph->data[tid + start_offset] = d_temp[tid];
}

extern "C"
void GpuArrayRearrangeX(BK_GPU::NeighbourGraph *graph,
    BK_GPU::GPU_Stack* stack,BK_GPU::GPU_CSR *InputGraph,unsigned int *darray,int start_offset,int end_offset,int countOnes,hipStream_t &stream)
{
  int numElements = end_offset - start_offset + 1;

  int* d_temp;

  gpuErrchk(hipMalloc(&d_temp,sizeof(int)*numElements));

  kernelRearrangeGatherX<<<ceil((double)numElements/128),128,0,stream>>>(darray,d_temp,start_offset,end_offset,countOnes,graph,stack);

  DEV_SYNC;

  KernelRearrangeScatterX<<<ceil((double)numElements/128),128,0,stream>>>(d_temp,start_offset,end_offset,graph);

  DEV_SYNC;

  gpuErrchk(hipFree(d_temp));
}
