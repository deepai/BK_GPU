#include "hip/hip_runtime.h"
#include "kernels.cuh"

/**This kernel is used to swap the values and bring the values having 1 in the darray towards the
 * start_offset and the values having 0 towards the end_offset
 * This Kernel acts as a gatherer.
 *
 * @param darray //array containing the prefixSum
 * @param d_temp //array to store the gathered values
 * @param start_offset //startOffset in the graph->data
 * @param end_offset //endoffset
 * @param graph //input graph
 * @param stack //input stack
 */
__global__
void kernelRearrangeGatherX(unsigned int *darray,unsigned int *d_temp,int start_offset,int end_offset,int countZeroes,unsigned *data,BK_GPU::GPU_Stack* stack)
{
	int tid=threadIdx.x + blockDim.x*blockIdx.x;

	//Exceeds limit hence return
	if(tid+start_offset > end_offset)
		return;

	//get the current prefixsum value
	unsigned currVal=darray[tid];

	//get the previous prefixSum value
	unsigned prevVal=(tid==0)?0:darray[tid-1];

	int destination; //store destination here

	/**If( nextVal - currVal ) == 1, indicates that tid+start_offset is a neighbour. Hence
	 * ,its destination will be start_offset + currVal - 1(currval indicates number of 1s obtained previously)
	 *
	 * else , it indicates the tid+start_offset is not a neighbour. hence currVal - tid+start_offset
	 * indicates number of 0s preceding it.
	 */

	if(currVal - prevVal == 1)
		destination = currVal - 1;
	else
		destination = end_offset - (countZeroes - (tid+1 - currVal)) - start_offset;

	//Copy the current Element before swapping
	unsigned currElement=data[tid+start_offset];

	d_temp[destination] = currElement;

}

/**
 * In this kernel elements in the d_temp array are copied back into the graph->data
 * starting from start_offset.
 *
 * @param d_temp //temporary storage
 * @param start_offset //start_offset in graph->data
 * @param end_offset //end_offset in graph->data
 * @param graph //graph
 */
__global__
void KernelRearrangeScatterX(unsigned *d_temp,int start_offset,int end_offset,unsigned *data)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if(tid+start_offset > end_offset)
    return;

  data[tid + start_offset] = d_temp[tid];
}

extern "C"
void GpuArrayRearrangeX(BK_GPU::NeighbourGraph *graph,
    BK_GPU::GPU_Stack* stack,BK_GPU::GPU_CSR *InputGraph,unsigned int *darray,int start_offset,int end_offset,int countZeros,hipStream_t &stream)
{
  int numElements = end_offset - start_offset + 1;

  if(numElements < 2)
	  return;

  unsigned* d_temp;

  CudaError(hipMalloc(&d_temp,sizeof(int)*numElements));

  kernelRearrangeGatherX<<<ceil((double)numElements/128),128,0,stream>>>(darray,d_temp,start_offset,end_offset,countZeros,graph->data,stack);

  CudaError(hipStreamSynchronize(stream));

  KernelRearrangeScatterX<<<ceil((double)numElements/128),128,0,stream>>>(d_temp,start_offset,end_offset,graph->data);

  CudaError(hipStreamSynchronize(stream));

  CudaError(hipFree(d_temp));
}
