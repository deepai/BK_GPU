#include "hip/hip_runtime.h"
#include "kernels.cuh"

/**This kernel is used to swap the values and bring the values having 1 in the darray towards the
 * start_offset and the values having 0 towards the end_offset
 * This Kernel acts as a gatherer.
 *
 * @param darray //array containing the prefixSum
 * @param d_temp //array to store the gathered values
 * @param start_offset //startOffset in the graph->data
 * @param end_offset //endoffset
 * @param graph //input graph
 * @param stack //input stack
 */
__global__
void kernelRearrangeGatherP(unsigned int *darray,int *d_temp,int start_offset,int end_offset,int countZeroes,int *data,BK_GPU::GPU_Stack* stack)
{
	int tid=threadIdx.x + blockDim.x*blockIdx.x;

	//Exceeds limit hence return
	if(tid+start_offset > end_offset)
		return;

	//get the current prefixsum value
	int currVal=darray[tid];

	//get the previous prefixSum value
    int prevVal=(tid==0)?0:darray[tid-1];

	int destination; //store destination here

	/**If( nextVal - currVal ) == 1, indicates that tid+start_offset is a neighbour. Hence
	 * ,its destination will be start_offset + currVal - 1(currval indicates number of 1s obtained previously)
	 *
	 * else , it indicates the tid+start_offset is not a neighbour. hence currVal - tid+start_offset
	 * indicates number of 0s preceding it.
	 */

	if(currVal - prevVal == 1)
		destination = currVal - 1;
	else
		destination = end_offset - (countZeroes - (tid+1 - currVal)) - start_offset;

	//Copy the current Element before swapping
	int currElement=data[tid+start_offset];

	d_temp[destination] = currElement;

}

/**
 * In this kernel elements in the d_temp array are copied back into the graph->data
 * starting from start_offset.
 *
 * @param d_temp //temporary storage
 * @param start_offset //start_offset in graph->data
 * @param end_offset //end_offset in graph->data
 * @param graph //graph
 */
__global__
void KernelRearrangeScatterP(int *d_temp,int start_offset,int end_offset,int *data)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;

	if(tid+start_offset > end_offset)
		return;

	data[tid + start_offset] = d_temp[tid];
}


extern "C"
void GpuArrayRearrangeP(BK_GPU::NeighbourGraph *graph,
		BK_GPU::GPU_Stack* stack,BK_GPU::GPU_CSR *InputGraph,unsigned int *darray,int start_offset,int end_offset,int countZeroes,hipStream_t &stream)
{
	int numElements = end_offset - start_offset + 1;

	int* d_temp;

	CudaError(hipMalloc(&d_temp,sizeof(int)*numElements));

	kernelRearrangeGatherP<<<ceil((double)numElements/128),128,0,stream>>>(darray,d_temp,start_offset,end_offset,countZeroes,graph->data,stack);

	CudaError(hipStreamSynchronize(stream));

	KernelRearrangeScatterP<<<ceil((double)numElements/128),128,0,stream>>>(d_temp,start_offset,end_offset,graph->data);

	CudaError(hipStreamSynchronize(stream));

	CudaError(hipFree(d_temp));
}
