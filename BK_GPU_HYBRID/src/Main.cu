#include <cstdio>
#include <cstdlib>
#include "Host/CsrGraph.h"
#include "utilities.h"
#include "Device/GPUStack.h"
#include "Device/NeighbourGraph.h"
#include "Device/GPUCSR.h"
#include "Device/StackElement.h"
#include "kernels/kernels.cuh"
#include "Host/BKInstance.h"
#include "moderngpu/moderngpu.cuh"
#include "moderngpu/util/mgpucontext.h"
#include <iostream>
#include <algorithm>
#include <omp.h>

#include "hipcub/hipcub.hpp"

//#include <hipcub/hipcub.hpp>

#define gc fgetc

inline int isSpaceChar(char c) {
	return (c == ' ' || c == '\n' || c == '\r' || c == ',');
}
inline int FAST_IO(FILE* fp) {
	char ch;
	int val = 0;
	ch = gc(fp);
	while (isSpaceChar(ch))
		ch = gc(fp);
	val = 0;
	while (!isSpaceChar(ch)) {
		val = (val * 10) + (ch - '0');
		ch = gc(fp);
	}
	return val;
}

debugger dbg;

bool isBigEndian() {
	unsigned int i = 1;
	char *c = (char*) &i;
	if (*c)
		return false;
	else
		return true;
}

struct listbyPsize
{
	int Psize;
	int index;

	bool operator < (const listbyPsize &rhs) const
	{
		return (Psize > rhs.Psize);
	}

};

int main(int argc, char * argv[]) {
	if (argc < 6) {
		printf(
				"Argument 1 should be path of the Input Matrix File.\n"
						"Argument 2 should be 0 for undirected and 1 for directed.\n"
						"Argument 3 should be 0 for 0 index-based or 1 index based.\n"
						"Argument 4 should be 0 for mtx format or 1 for normal format.\n"
						"Argument 5 should be #Threads\n");
		exit(1);
	}

	FILE* fp = fopen(argv[1], "r");

	bool undirected = atoi(argv[2]);
	bool oneIndexBased = atoi(argv[3]);
	bool nonmtxFormat = atoi(argv[4]);
	int numThreads = atoi(argv[5]);

	omp_set_num_threads(numThreads);

	int N, E;
	int a, b;

	char c;

	if (!nonmtxFormat)
		while ((c = fgetc(fp)) != '\n')
			;

	if (!nonmtxFormat)
		N = FAST_IO(fp);
	N = FAST_IO(fp);
	E = FAST_IO(fp);

	Graph *g1 = new Graph(N, (!undirected) ? 2 * E : E);

	printf("Edges= %d,Nodes= %d\n", g1->Edges, g1->Nodes);

	for (int i = 0; i < E; i++) {
		a = FAST_IO(fp);
		b = FAST_IO(fp);
		if (!oneIndexBased)
			g1->insertEdges(a - 1, b - 1, undirected);
		else
			g1->insertEdges(a, b, undirected);

	}

	CudaError(hipDeviceReset());

	g1->sortEdgeLists();

	g1->calculateKores(isBigEndian()); //Added

	g1->calculateNeighbourArray();

	int Core =
			g1->KCoreValues[g1->neighbourArray[g1->neighbourArray.size() - 1][0]];
	int loc = g1->neighbourArray.size() - 1;

	printf("CoreSize = %d\n", Core);

	int totalSize = 0;
	int countNodes = 0;

	for (int i = g1->neighbourArray.size() - 1; i >= 0; i--) {

		int lastindex = g1->neighbourArray[i].size()-1;

		if (g1->KCoreValues[g1->neighbourArray[i][lastindex]] == Core) {
			loc = i;
			countNodes++;
			totalSize += g1->neighbourArray[loc].size();
			totalSize += g1->preDegeneracyVertices[loc].size();
			//printf("%d \n",totalSize);
		} else
			break;
	}

	BK_GPU::NeighbourGraph *Ng = new BK_GPU::NeighbourGraph(countNodes,
			totalSize);

	printf("Number of Elements = %d,totalSize = %d\n", countNodes, totalSize);

	int offset = 0;

	int nodeIndex = 0;
	int countofStack = countNodes;

	BK_GPU::GPU_Stack **stack;

	//hipMallocManaged(&stack, sizeof(BK_GPU::GPU_Stack*) * countofStack);

	stack=new BK_GPU::GPU_Stack*[countofStack];

	DEV_SYNC;

	//Count of stack = Count Nodes which has Corenumber as Core
	//

/**
This L array is used to first sort the neighbour array values by Psize
**/
	std::vector<listbyPsize> L(countNodes);

	for (int i = loc; i < g1->neighbourArray.size(); i++) {

//Calculate Psize
		int Psize = g1->neighbourArray[i].size();
//Calculate Rsize
		int Rsize = g1->preDegeneracyVertices[i].size();

//Stack Node
		stack[nodeIndex] = new BK_GPU::GPU_Stack(Psize);

		L[nodeIndex].Psize = Psize-1;
		L[nodeIndex].index = nodeIndex;

		Ng->copy(nodeIndex++, offset, g1->neighbourArray[i].data(),Psize,g1->preDegeneracyVertices[i].data(),Rsize);

		BK_GPU::StackElement *element=new BK_GPU::StackElement(offset,Rsize ,offset + Rsize, Psize - 1 , offset + Rsize + Psize - 1, 1,
				0,g1->neighbourArray[i][Psize-1], true);

		stack[nodeIndex-1]->push(element);

		offset += (Psize + Rsize);

		delete element;
	}

//Sort the graph by currP size
	std::sort(L.begin(),L.end());

	int Cliquesize = Ng->cliqueSize;

	//Copy the Input graph in CSR format to the GPU
	BK_GPU::GPU_CSR *gpuGraph = new BK_GPU::GPU_CSR(*g1);

//	for(int i=0;i<countNodes;i++)
//	{
//		int idx = L[i].index;
//		std::cout <<"Psize is :" << L[i].Psize << std::endl;
//		for(int j=Ng->dataOffset[idx];j<Ng->dataOffset[idx+1];j++)
//			std::cout <<  Ng->data[j] + 1 << " ";
//		std::cout << std::endl;
//	}

	//Create required number of cudaStreams
	hipStream_t stream[numThreads];

	std::cout<<omp_get_num_threads()<<std::endl;

	for(int i=0;i<numThreads;i++)
		hipStreamCreate(&(stream[i]));

	//Create required number of ContextPointers
	mgpu::ContextPtr *Contextptr=new mgpu::ContextPtr[numThreads];
	for(int i=0;i<numThreads;i++)
		Contextptr[i]=mgpu::CreateCudaDeviceAttachStream(stream[i]);

	//MultiThreaded Application
	//#pragma omp parallel for
	for(int i=0;i<L.size();i++)
	{

		//ThreadId of each omp thread starting from 0.
		int threadIdx=omp_get_thread_num();
		//printf("tid is %d\n",tid);

		//Instance variable reference. Instance variable is responsible to find Cliques starting with a vertex.
		BK_GPU::BKInstance *instance;

		//Make an object corresponding to the instance.
		instance=new BK_GPU::BKInstance(g1,gpuGraph,Ng,stack[L[i].index],Contextptr,numThreads,0);

		//Invoke the RunCliqueFinder Method.
		instance->RunCliqueFinder(i);

		//Wait till all resources are freed within the stream.
		hipStreamSynchronize(stream[0]);

		//make the reference empty.
		delete instance;
	}

	//Destroy the streams.
	for(int i=0;i<numThreads;i++)
	{
			hipStreamDestroy(stream[i]);
			Contextptr[i]->Release();
	}

	//hipStreamDestroy(stream[0]);

	delete gpuGraph;
	delete Ng;

	//debug("hello");
	fclose(fp);

	return 0;
}

