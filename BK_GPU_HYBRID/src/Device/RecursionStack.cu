/*
 * RecursionStack.cpp
 *
 *  Created on: 16-Oct-2015
 *      Author: debarshi
 */

#include "RecursionStack.h"

namespace BK_GPU {

RecursionStack::RecursionStack(int size,hipStream_t &stream) {
	// TODO Auto-generated constructor stub
	top=0;
	CudaError(hipMalloc(&elements,sizeof(int)*size));

}

RecursionStack::~RecursionStack()
{
	CudaError(hipFree(this->elements));
}

} /* namespace BK_GPU */
