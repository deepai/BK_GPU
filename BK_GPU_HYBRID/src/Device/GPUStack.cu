/*
 * GPUStack.cpp
 *
 *  Created on: 09-Sep-2015
 *      Author: debarshi
 */

#include "GPUStack.h"

namespace BK_GPU {


GPU_Stack::GPU_Stack(int size) {
	// TODO Auto-generated constructor stub
	top = 0;
	maxCliqueSize = size;
	CudaError(hipMalloc(&elements, sizeof(StackElement) * (size+1)));

}


GPU_Stack::~GPU_Stack() {
	// TODO Auto-generated destructor stub
	CudaError(hipFree(elements));
}


} /* namespace BK_GPU */
