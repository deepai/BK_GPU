/*
 * GPUCSR.cpp
 *
 *  Created on: 14-Sep-2015
 *      Author: debarshi
 */

#include "GPUCSR.h"

namespace BK_GPU {

GPU_CSR::GPU_CSR(Graph &graph) {
	// TODO Auto-generated constructor stub
	this->Nodes = graph.Nodes;
	this->Edges = graph.Edges;

	//copy from host memory to device_memory
	gpuErrchk(
			hipMallocManaged(&(this->rowOffsets),
					sizeof(int) * (this->Nodes + 1)));
	gpuErrchk(
			hipMemcpy(this->rowOffsets, graph.rowOffset.data(),
					sizeof(unsigned) * (this->Nodes + 1),
					hipMemcpyHostToDevice));
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipMallocManaged(&(this->Columns), sizeof(int) * (this->Edges)));

	gpuErrchk(
			hipMemcpy(this->Columns, graph.columns.data(),
					sizeof(unsigned) * (this->Edges), hipMemcpyHostToDevice));

	gpuErrchk(hipDeviceSynchronize());

}

void *GPU_CSR::operator new(size_t len) {
	void *ptr;
	gpuErrchk(hipMallocManaged(&ptr, len * sizeof(BK_GPU::GPU_CSR)));
	gpuErrchk(hipDeviceSynchronize());
	return ptr;
}

void GPU_CSR::operator delete(void *ptr) {
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipFree(ptr));
}

} /* namespace BK_GPU */
