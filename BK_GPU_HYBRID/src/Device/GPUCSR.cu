/*
 * GPUCSR.cpp
 *
 *  Created on: 14-Sep-2015
 *      Author: debarshi
 */

#include "GPUCSR.h"

namespace BK_GPU {

GPU_CSR::~GPU_CSR()
{
	CudaError(hipFree(this->rowOffsets));
	CudaError(hipFree(this->Columns));
}

GPU_CSR::GPU_CSR(Graph &graph) {
	// TODO Auto-generated constructor stub
	this->Nodes = graph.Nodes;
	this->Edges = graph.Edges;

	//copy from host memory to device_memory
	CudaError(
			hipMalloc(&(this->rowOffsets),
					sizeof(int) * (this->Nodes + 1)));
	CudaError(
			hipMemcpy(this->rowOffsets, graph.rowOffset.data(),
					sizeof(unsigned) * (this->Nodes + 1),
					hipMemcpyHostToDevice));
	DEV_SYNC;

	CudaError(hipMalloc(&(this->Columns), sizeof(int) * (this->Edges)));

	CudaError(
			hipMemcpy(this->Columns, graph.columns.data(),
					sizeof(unsigned) * (this->Edges), hipMemcpyHostToDevice));

	DEV_SYNC;

}

} /* namespace BK_GPU */
